#include <iostream>
#include <fstream>
#include "stdio.h"
#include <vector>
#include <hip/hip_runtime.h>
#include "math.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/extrema.h>

#include "cuPrintf.cu"

#include "TROOT.h"
#include "TApplication.h"
#include "TCanvas.h"
#include "TGraphErrors.h"
#include "TMultiGraph.h"
#include "TLegend.h"
#include "TPaveText.h"
#include "TStopwatch.h"
#include "TMath.h"
#include "TAxis.h"

#include "AhHoughTransformation.h"

/**
 * @brief Reads data from specific formated file into three given vectors
 * @param filename       Filename (include path if necessary)
 * @param x              x-value vector (1st row of file)
 * @param y              y-value vector (2nd row of file)
 * @param r              r-value vector (4th row of file)
 * @param upToLineNumber Up to with line (including that line) the file will be read. Starts at line 1.
 *
 * Uses just x, y and r at the moment, because that's all I need.
 */
void readPoints(std::string filename, std::vector<double> &x, std::vector<double> &y, std::vector<double> &r, std::vector<int> &vI, int upToLineNumber = 2) {
	std::ifstream file(filename.c_str());
	float tempX, tempY, tempZ, tempR, tempI;
	char tempChar[10];
	int i = 1;
	if (!file.good() || file.fail()) std::cout << "Failed opening file!" << std::endl;
	while(i <= upToLineNumber && file >> tempX >> tempY >> tempZ >> tempR >> tempI >> tempChar) {
		x.push_back(tempX);
		y.push_back(tempY);
		r.push_back(tempR);
		vI.push_back((int)tempI);
		i++;
	}
	file.close();
}

int main (int argc, char** argv) {
	int verbose = 1;
	
	//! fill original data
	std::vector<double> x;
	std::vector<double> y;
	std::vector<double> r;
	std::vector<int> vI;
	// readPoints("data.dat", x, y, r, 18);
	readPoints("real_data.txt", x, y, r, vI, 10000);
	
	thrust::host_vector<double> h_x, h_xMid, h_xMax;
	thrust::host_vector<double> h_y, h_yMid, h_yMax;
	thrust::host_vector<double> h_r, h_rMid, h_rMax;

	//! Setting parameters
	double maxAngle = 180*2;

	// ##################
	// ### FIRST PART ###
	// ##################
	
	//! Change container from std::vector to thrust::host_vector
	int singleRange = 1;
	int midRangeMax = 100;
	int maxRangeMax = 999;
	for (int i = 0; i < x.size(); ++i) {
		if (vI[i] <= maxRangeMax) {
			h_xMax.push_back(x[i]);
			h_yMax.push_back(y[i]);
			h_rMax.push_back(r[i]);
			if (vI[i] <= midRangeMax) {
				h_xMid.push_back(x[i]);
				h_yMid.push_back(y[i]);
				h_rMid.push_back(r[i]);
				if (vI[i] <= singleRange) {
					std::cout << x[i] << std::endl;
					h_x.push_back(x[i]);
					h_y.push_back(y[i]);
					h_r.push_back(r[i]);
				}
			}
		}
	}

	std::vector<double> timesConfMap, timesGenAngles, timesHoughTrans, timesAll, timesAllMid, timesAllMax;

// int currentEvent = 0;
// int nEvents = 10;
// for (int iEvents = 1; iEvents <= nEvents; ++iEvents) {
// 	while (vI[currentEvent] == iEvents) {
// 		currentEvent++;
// 	}
// }
	std::vector<double> vOfDegreeCellsizes;
	// vOfDegreeCellsizes.push_back(60);
	vOfDegreeCellsizes.push_back(30);
	// vOfDegreeCellsizes.push_back(25);
	vOfDegreeCellsizes.push_back(20);
	vOfDegreeCellsizes.push_back(15);
	vOfDegreeCellsizes.push_back(10);
	vOfDegreeCellsizes.push_back(5);
	vOfDegreeCellsizes.push_back(2);
	vOfDegreeCellsizes.push_back(1);
	vOfDegreeCellsizes.push_back(0.5);
	for (int i = 0; i < vOfDegreeCellsizes.size(); ++i) {
		AhHoughTransformation * houghTrans = new AhHoughTransformation(h_x, h_y, h_r, maxAngle, vOfDegreeCellsizes[i], true);
		AhHoughTransformation * houghTransMid = new AhHoughTransformation(h_xMid, h_yMid, h_rMid, maxAngle, vOfDegreeCellsizes[i], true);
		AhHoughTransformation * houghTransMax = new AhHoughTransformation(h_xMax, h_yMax, h_rMax, maxAngle, vOfDegreeCellsizes[i], true);

		timesConfMap.push_back(houghTrans->GetTimeConfMap());
		timesGenAngles.push_back(houghTrans->GetTimeGenAngles());
		timesHoughTrans.push_back(houghTrans->GetTimeHoughTransform());
		timesAll.push_back(houghTrans->GetTimeConfMap() + houghTrans->GetTimeGenAngles() + houghTrans->GetTimeHoughTransform());

		timesAllMid.push_back(houghTransMid->GetTimeConfMap() + houghTransMid->GetTimeGenAngles() + houghTransMid->GetTimeHoughTransform());
		timesAllMax.push_back(houghTransMax->GetTimeConfMap() + houghTransMax->GetTimeGenAngles() + houghTransMax->GetTimeHoughTransform());
		delete houghTrans;
		delete houghTransMid;
		delete houghTransMax;
	}

	TGraphErrors * graphConfMap = new TGraphErrors();
	TGraphErrors * graphGenAngles = new TGraphErrors();
	TGraphErrors * graphHoughTrans = new TGraphErrors();
	TGraphErrors * graphAll = new TGraphErrors();
	TGraphErrors * graphAllMid = new TGraphErrors();
	TGraphErrors * graphAllMax = new TGraphErrors();

	for (int i = 0; i < timesAll.size(); ++i) {
		graphConfMap->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesConfMap[i]);
		graphGenAngles->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesGenAngles[i]);
		graphHoughTrans->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesHoughTrans[i]);
		graphAll->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesAll[i]);
		// graphAllMid->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesAllMid[i]/midRangeMax);
		// graphAllMax->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesAllMax[i]/maxRangeMax);
		graphAllMid->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesAllMid[i]);
		graphAllMax->SetPoint(i, maxAngle/vOfDegreeCellsizes[i], timesAllMax[i]);
		std::cout << "i = " << i << ":: time per Event MID: " << timesAllMid[i]/midRangeMax << " ---- time per Event MAX: " << timesAllMax[i]/maxRangeMax << std::endl;

	}
	int dotSize = 1;

	graphConfMap->SetLineColor(kOrange);
	graphConfMap->SetFillColor(kYellow - 10);
	graphConfMap->SetMarkerStyle(kFullTriangleUp);
	graphConfMap->SetMarkerSize(dotSize);
	graphConfMap->SetMarkerColor(graphConfMap->GetLineColor() + 2);
	graphConfMap->SetTitle("Conf Map");
	graphGenAngles->SetLineColor(kOrange+6);
	graphGenAngles->SetFillColor(kYellow - 9);
	graphGenAngles->SetMarkerStyle(kFullSquare);
	graphGenAngles->SetMarkerSize(dotSize);
	graphGenAngles->SetMarkerColor(graphGenAngles->GetLineColor() +2);
	graphGenAngles->SetTitle("Gen Angles");
	graphHoughTrans->SetLineColor(kRed);
	graphHoughTrans->SetFillColor(graphHoughTrans->GetLineColor() - 9);
	graphHoughTrans->SetMarkerStyle(kFullTriangleDown);
	graphHoughTrans->SetMarkerSize(dotSize);
	graphHoughTrans->SetMarkerColor(graphHoughTrans->GetLineColor() +2);
	graphHoughTrans->SetTitle("Hough Trans");
	graphAll->SetLineColor(kBlue);
	graphAll->SetFillColor(graphAll->GetLineColor() - 9);
	graphAll->SetMarkerStyle(kFullDotLarge);
	graphAll->SetMarkerSize(dotSize);
	graphAll->SetMarkerColor(graphAll->GetLineColor() +2);
	graphAll->SetTitle("All (1 Evt)");


	graphAllMid->SetLineColor(kCyan);
	graphAllMid->SetFillColor(graphAllMid->GetLineColor() - 9);
	graphAllMid->SetMarkerStyle(kFullTriangleUp);
	graphAllMid->SetMarkerSize(dotSize);
	graphAllMid->SetMarkerColor(graphAllMid->GetLineColor() +2);
	graphAllMid->SetTitle("100 Events");
	graphAllMax->SetLineColor(kGreen);
	graphAllMax->SetFillColor(graphAllMax->GetLineColor() - 9);
	graphAllMax->SetMarkerStyle(kFullTriangleDown);
	graphAllMax->SetMarkerSize(dotSize);
	graphAllMax->SetMarkerColor(graphAllMax->GetLineColor() +2);
	graphAllMax->SetTitle("1000 Events");

	TApplication *theApp = new TApplication("app", &argc, argv, 0, -1);
	TCanvas * c1 = new TCanvas("c1", "default", 0, 0, 800, 600);
	
	TMultiGraph * mg = new TMultiGraph();

	mg->Add(graphAll);
	mg->Add(graphHoughTrans);
	mg->Add(graphGenAngles);
	mg->Add(graphConfMap);
	
	mg->Draw("AP");
	mg->GetXaxis()->SetTitle("Number of grid points/#");
	mg->GetYaxis()->SetTitle("Time/ms");
	mg->GetYaxis()->SetTitleOffset(1.4);
	
// 	gPad->SetLogy();
	
	TLegend * leg = c1->BuildLegend(0.7,0.8,0.95,0.95);
	leg->SetFillColor(kWhite);
	// TPaveText * formulaLeg = new TPaveText(0.35,0.75,0.45,0.9,"blNDC");
	// formulaLeg->AddText("#sum^{N}_{i} = x^{2}");
	// formulaLeg->SetFillColor(kWhite);
	// formulaLeg->SetBorderSize(1);
	// formulaLeg->SetTextSize(formulaLeg->GetTextSize()*0.7);
	// formulaLeg->Draw();
	c1->Update();

	// ###################
	// ### SECOND PART ###
	// ###################
	TCanvas * c2 = new TCanvas("c2","default2",300,0,800,600);
	TMultiGraph * mgEvents = new TMultiGraph();
	
	graphAll->SetTitle("1 Event");
	mgEvents->Add(graphAll);
	mgEvents->Add(graphAllMid);
	mgEvents->Add(graphAllMax);

	mgEvents->Draw("AP");

	TLegend * legEvents = c2->BuildLegend(0.7,0.8,0.95,0.95);
	legEvents->SetFillColor(kWhite);
	c2->SetLogy();
	mgEvents->GetXaxis()->SetTitle("Hough angle #alpha resolution");
	// mgEvents->GetYaxis()->SetTitle("Computation Time per Event/ms");
	mgEvents->GetYaxis()->SetTitle("Computation time/ms");
	mgEvents->GetYaxis()->SetTitleOffset(1.3);
	c2->Update();

	c2->Print("ht_runtime_diff_events.pdf");
	c1->Print("ht_runtime_diff_stages_of_chain.pdf");
	theApp->Run();

}
