#include "AhHoughTransformation.h"

AhHoughTransformation::AhHoughTransformation()
{}

AhHoughTransformation::AhHoughTransformation(thrust::host_vector<TYPE> xValues, thrust::host_vector<TYPE> yValues, TYPE maxAngle, TYPE everyXDegrees, bool doTiming)
: fXValues(xValues),
  fYValues(yValues),
  fMaxAngle(maxAngle),
  fEveryXDegrees(everyXDegrees),
  fDoTiming(doTiming),
  fUseIsochrones(false)
{
	DoEverything();
}

AhHoughTransformation::AhHoughTransformation(thrust::host_vector<TYPE> xValues, thrust::host_vector<TYPE> yValues, thrust::host_vector<TYPE> rValues, TYPE maxAngle, TYPE everyXDegrees, bool doTiming)
: fXValues(xValues),
  fYValues(yValues),
  fRValues(rValues),
  fMaxAngle(maxAngle),
  fEveryXDegrees(everyXDegrees),
  fDoTiming(doTiming),
  fUseIsochrones(true)
{
	// std::cout << "fXValues.size() = " << fXValues.size() << ", fYValues.size()" << fYValues.size() << "fRValues.size() = " << fRValues.size() << ", fMaxAngle = " << fMaxAngle << ", fEveryXDegrees = " << fEveryXDegrees << ", fUseIsochrones = " << fUseIsochrones << std::endl;
	DoEverything();
}

AhHoughTransformation::~AhHoughTransformation()
{}

void AhHoughTransformation::EventTiming_start() {
	hipEventCreate(&fEventStart);
	hipEventCreate(&fEventStop);
	hipDeviceSynchronize();
	hipEventRecord(fEventStart, 0);
}

float AhHoughTransformation::EventTiming_stop() {
	float currentTime;
	hipEventRecord(fEventStop, 0);
	hipEventSynchronize(fEventStop);
	hipEventElapsedTime(&currentTime, fEventStart, fEventStop);
	hipEventDestroy(fEventStart);
	hipEventDestroy(fEventStop);

	return currentTime;
}

void AhHoughTransformation::DoEverything() {
	if (true == fDoTiming) EventTiming_start();
	DoConformalMapping();
	if (true == fDoTiming) fTimeConfMap = EventTiming_stop();

	if (true == fDoTiming) EventTiming_start();
	DoGenerateAngles();
	if (true == fDoTiming) fTimeGenAngles = EventTiming_stop();

	if (true == fDoTiming) EventTiming_start();
	DoHoughTransform();
	if (true == fDoTiming) fTimeHoughTransform = EventTiming_stop();
}

void AhHoughTransformation::DoConformalMapping() {
	//! conformal mapping

	fCXValues.resize(fXValues.size());
	fCYValues.resize(fYValues.size());

	ConformalMapOneVector(fXValues, fCXValues);
	ConformalMapOneVector(fYValues, fCYValues);

	if (true == fUseIsochrones) {
		fCRValues.resize(fRValues.size());

		ConformalMapOneVector(fRValues, fCRValues);
	}
}

void AhHoughTransformation::ConformalMapOneVector(thrust::device_vector<TYPE> &originalData, thrust::device_vector<TYPE> &mappedData) {
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				originalData.begin(),
				fXValues.begin(),
				fYValues.begin()
			)
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				originalData.end(),
				fXValues.end(),
				fYValues.end()
			)
		),
		mappedData.begin(),
		my::confMap()
	);
	std::cout << "--DEBUG-- AhHoughTransformation::ConformalMapOneVector ### Been here!" << std::endl;
}

void AhHoughTransformation::DoGenerateAngles() {
	fAngles.resize(fMaxAngle/fEveryXDegrees); //!< Resize angle vector to match the actual size
	//! Fill angle vector with angles in appropriate stepping
	TYPE zero = 0.;
	thrust::sequence(
		fAngles.begin(), 
		fAngles.end(), 
		zero, 
		fEveryXDegrees
	);
}

template <class T>
void AhHoughTransformation::DoHoughTransformOnePoint(thrust::constant_iterator<T> data, thrust::device_vector<TYPE> &d_tempData) {
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				fAngles.begin(),
				data
			)
		),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				fAngles.end(),
				data
			)
		),
		d_tempData.begin(),
		my::htransf()
	);
}

void AhHoughTransformation::DoHoughTransform() {
	/** Attention
	* For every (x*,y*) point a hough transform is done!
	* While this hough transform itself is done in parallel, all hough transforms in whole are done serial, one by one after an other.
	* This might be a point of huge improvements
	*/
	for (int iDataPoints = 0; iDataPoints < fXValues.size(); iDataPoints++) {
		thrust::device_vector<TYPE> d_tempData(fAngles.size()); //!< Temp vector which is being filled and then pushed back to the main return vector. For every angle point theres a data point, so that's the size of it

		if (false == fUseIsochrones) {
			thrust::constant_iterator<thrust::tuple<TYPE, TYPE> > currentData(thrust::make_tuple(fCXValues[iDataPoints], fCYValues[iDataPoints])); //!< create constant iterator for the conf mapped data 2-tuples

			//! following transformation uses the operator of htransf to run over all elements
			//!   elements being a iterator from angles.start to angles.end with each time the constant iterator with the 	conf mapped 2-tuple
			//!   the result of the calculation is written in to the d_tempData vector
			DoHoughTransformOnePoint<thrust::tuple<TYPE, TYPE> >(currentData, d_tempData);
			// thrust::transform(
			// 	thrust::make_zip_iterator(
			// 		thrust::make_tuple(
			// 			fAngles.begin(),
			// 			currentData
			// 		)
			// 	),
			// 	thrust::make_zip_iterator(
			// 		thrust::make_tuple(
			// 			fAngles.end(),
			// 			currentData
			// 		)
			// 	),
			// 	d_tempData.begin(),
			// 	my::htransf()
			// );
		} else { // (true == fUseIsochrones)
			thrust::constant_iterator<thrust::tuple<TYPE, TYPE, TYPE> > currentData(thrust::make_tuple(fCXValues[iDataPoints], fCYValues[iDataPoints], fCRValues[iDataPoints]));
			//! following transformation uses the operator of htransf to run over all elements
			//!   elements being a iterator from angles.start to angles.end with each time the constant iterator with the 	conf mapped 2-tuple
			//!   the result of the calculation is written in to the d_tempData vector
			DoHoughTransformOnePoint<thrust::tuple<TYPE, TYPE, TYPE> >(currentData, d_tempData);
			// thrust::transform(
			// 	thrust::make_zip_iterator(
			// 		thrust::make_tuple(
			// 			fAngles.begin(),
			// 			currentData
			// 		)
			// 	),
			// 	thrust::make_zip_iterator(
			// 		thrust::make_tuple(
			// 			fAngles.end(),
			// 			currentData
			// 		)
			// 	),
			// 	d_tempData.begin(),
			// 	my::htransf()
			// );
		}
		
		
		fTransformedPoints.push_back(d_tempData); //!< push it back to the main data stack vector

	}
}
