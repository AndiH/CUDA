#include "hip/hip_runtime.h"
// The following compiler flag will change the default usage of floats in this class to doubles
// Use, e.g., as follows: g++ hough.cpp AhHoughTransformation.o -DUSE_DOUBLES=1 
#ifdef USE_DOUBLES
	typedef double TYPE;
#else
	typedef float TYPE;
#endif

#include <iostream>
#include <fstream>
#include <algorithm> // to find minimum --> min_element
#include "stdio.h"
#include <vector>
#include <hip/hip_runtime.h>
#include "math.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/extrema.h>

#include "TH2D.h"
#include "TROOT.h"
#include "TApplication.h"
#include "TMatrixD.h"
#include "TCanvas.h"
#include "TStopwatch.h"
#include "TStyle.h"

#include <cusp/coo_matrix.h>
#include <cusp/print.h>

#include "AhTwoArraysToMatrix.h"
#include "AhHoughTransformation.h"


/**
 * @mainpage Conformal Mapping and Hough Transformation in CUDA Thrust
 * 
 * <h1>HoughTrust</h1>
 * <h2>A PhD project</h2>
 * 
 * Using CUDA Thrust to conformal map and hough transform points of a tracker. Two dimensionally.
 * 
 * @author Andreas Herten
 */

/**
 * @file houghtransform.cu
 * 
 * @brief Everything is in that one file
 * 
 * Sorry for that, but if/when I'm going to need any special functions, I will extract them
 **/

/**
 * @brief Reads data from specific formated file into three given vectors
 * @param filename       Filename (include path if necessary)
 * @param x              x-value vector (1st row of file)
 * @param y              y-value vector (2nd row of file)
 * @param r              r-value vector (4th row of file)
 * @param upToLineNumber Up to with line (including that line) the file will be read. Starts at line 1.
 *
 * Uses just x, y and r at the moment, because that's all I need.
 */
void readPoints(std::string filename, std::vector<TYPE> &x, std::vector<TYPE> &y, std::vector<TYPE> &r, int upToLineNumber = 2) {
	std::ifstream file(filename.c_str());
	float tempX, tempY, tempZ, tempR, tempI;
	char tempChar[10];
	int i = 1;
	if (!file.good() || file.fail()) std::cout << "Failed opening file!" << std::endl;
	while(i <= upToLineNumber && file >> tempX >> tempY >> tempZ >> tempR >> tempI >> tempChar) {
		x.push_back(tempX);
		y.push_back(tempY);
		r.push_back(tempR);
		i++;
	}
	file.close();
}

/**
 * @brief Helper function which simply prints out the contents of a tuple
 * @param thatTuple A tuple of doubles to be printed
 * @return Nothing, it's a void.
 */
void printTuple (thrust::tuple<TYPE, TYPE> thatTuple) {
	std::cout << thrust::get<0>(thatTuple) << " - " << thrust::get<1>(thatTuple);
}

/**
 * @brief Prints out every element of a vector
 * @param Vector - and a type (this method is a template)
 */
template <class T>
void printVector (const T & v) {
	for (int i = 0; i < v.size(); ++i) {
		std::cout << v[i] << " ";
	}
	std::cout << std::endl;
}

/**
 * @brief Adds up TH2D histograms
 * @param vHistograms A std vector with pointers to TH2D histograms
 * @return A pointer to a TH2D histogram
 */
TH2D * addVectorOfPToHistograms (std::vector<TH2D* > vHistograms) {
	TH2D * tempHist = new TH2D(*(vHistograms[0]));
	for (int i = 1; i < vHistograms.size(); i++) {
		tempHist->Add(vHistograms[i]);
	}
	return tempHist;
}

int main (int argc, char** argv) {
	gStyle->SetOptStat(0);
	int verbose = 1;
	
	//! fill original data
	std::vector<TYPE> x;
	std::vector<TYPE> y;
	std::vector<TYPE> r;
	// readPoints("data.dat", x, y, r, 18);

	std::string inputFileName = "real_data.txt";
	int readInUpToLineNumber = 20;
	if (argc > 2) readInUpToLineNumber = static_cast<int>(atof(argv[2]));
	if (argc > 3) inputFileName = argv[3];
	readPoints(inputFileName, x, y, r, readInUpToLineNumber);

	//! Change container from std::vector to thrust::host_vector
	thrust::host_vector<TYPE> h_x = x;
	thrust::host_vector<TYPE> h_y = y;
	thrust::host_vector<TYPE> h_r = r;
	
	TStopwatch myWatch;

	//! Setting parameters
	TYPE maxAngle = 180; //!< Hough transform ranges from 0 deg to 180 deg
	TYPE everyXDegrees = 30; //!< make a point every X degrees of alpha; default = 30
	if (argc > 1) everyXDegrees = (TYPE)atof(argv[1]); //!< overwrite default value to what was given by command line

	//! Simple (x,y) coordinates
	AhHoughTransformation * houghTrans = new AhHoughTransformation(h_x, h_y, maxAngle, everyXDegrees, true);

	//! Use isochrones - (x,y,r) coordinates
	// maxAngle *= 2; //!< for isochrones, hough transformation goes from 0 to 360
	// AhHoughTransformation * houghTrans = new AhHoughTransformation(h_x, h_y, h_r, maxAngle, everyXDegrees, true);

	thrust::device_vector<TYPE> alphas = houghTrans->GetAngles();
	std::vector<thrust::device_vector<TYPE> > transformedPoints = houghTrans->GetVectorOfTransformedPoints();

	// std::cout << "It took " << houghTrans->GetTimeHoughTransform()/1000 << "s for actual HT." << std::endl;

	/*
	 * ### Make CUSP Matrix ###
	 */
	//! Find upper and lower borders of histograms
	int nBinsX = (int) maxAngle/everyXDegrees;
	TYPE minValueX = 0;
	TYPE maxValueX = maxAngle;
	if (verbose > 0) std::cout << "nBinsX = " << nBinsX << ", minValueX = " << minValueX << ", maxValueX = " << maxValueX << std::endl;

	int nBinsY = maxAngle/everyXDegrees;
	// TYPE minValueY = -0.4;
	// TYPE maxValueY = 0.7;
	// if (verbose > 0) std::cout << "nBinsY = " << nBinsY << ", minValueY = " << minValueY << ", maxValueY " << maxValueY << std::endl;
	//! Automatically get y borders
	std::cout << transformedPoints.size() << std::endl;
	TYPE minValueY = transformedPoints[0][0];
	TYPE maxValueY = transformedPoints[0][0];
	std::cout << "minValueY = " << minValueY << ", maxValueY = " << maxValueY << std::endl;
	for (int i = 0; i < transformedPoints.size(); i++) {
		thrust::device_vector<TYPE> tempD(transformedPoints[i]);
		TYPE minimum = *(thrust::min_element(tempD.begin(), tempD.end()));
		TYPE maximum = *(thrust::max_element(tempD.begin(), tempD.end()));
		if (minimum < minValueY) minValueY = minimum;
		if (maximum > maxValueY) maxValueY = maximum;
		std::cout << "minimum = " << minimum << ", maximum = " << maximum << std::endl;
	}
	minValueY *= 1.1; //!< make edges a little smoother
	maxValueY *= 1.1; //!< make edges a little smoother
	std::cout << "minValueY = " << minValueY << ", maxValueY = " << maxValueY << std::endl;
	
	//! Create matrices
	std::vector<TH2D*> theHistograms;
	std::vector< cusp::coo_matrix<int, TYPE, cusp::device_memory> > theMatrices;
	std::vector<AhTwoArraysToMatrix> theObjects;
	for (int i = 0; i < transformedPoints.size(); i++) {
		AhTwoArraysToMatrix tempObject(
			thrust::device_vector<TYPE> (alphas), thrust::device_vector<TYPE> (transformedPoints[i]),
			nBinsX,
			minValueX,
			maxValueX,
			nBinsY,
			minValueY,
			maxValueY,
			true,
			true
		);

		if (verbose > 0) std::cout << "Some matrix parameters: " << tempObject.GetNBinsX() << " " << tempObject.GetXlow() << " " << tempObject.GetXup() << " "<< tempObject.GetNBinsY() <<  " " << tempObject.GetYlow() << " " << tempObject.GetYup() << std::endl;

		theHistograms.push_back(tempObject.GetHistogram());
		theMatrices.push_back(tempObject.GetCUSPMatrix());
		theObjects.push_back(tempObject);
		char tempchar[5];
		sprintf(tempchar, "%d", i);
		theHistograms[i]->SetName(tempchar);
	}


	/*
	 * ### DEBUG Output ###
	 */
	if (verbose > 5) cusp::print(theMatrices[0]);

	if (verbose > 0) { // Timings
		for (int i = 0; i < theObjects.size(); i++) {
			std::cout << "Timings for histogram " << i << std::endl;
			std::cout << "  T for translating values: " << theObjects[i].GetTimeTranslateValues() << std::endl;
			std::cout << "  T for sorting histogram vectors: " << theObjects[i].GetTimeHistSort() << std::endl;
			std::cout << "  T for summing histogram vectors: " << theObjects[i].GetTimeHistSum() << std::endl;
			std::cout << "  T for generating Matrix: " << theObjects[i].GetTimeCreateTMatrixD() << std::endl;
			std::cout << "  T for generating TH2D: " << theObjects[i].GetTimeCreateTH2D() << std::endl;
		}
	}
	
	if (verbose > 1) {
	for (int i = 0; i < transformedPoints.size(); i++) {
		std::cout << "transformedPoints[" << i << "].size() = " << transformedPoints[i].size() << std::endl;
		for (int j = 0; j < transformedPoints[i].size(); j++) {
			std::cout << "  transformedPoints[" << i << "][" << j << "] = " << transformedPoints[i][j] << std::endl;
		}
	}
	}
	
	myWatch.Stop();
	std::cout << "For operations, it took me ";
	myWatch.Print();
	std::cout << std::endl;
	
	/*
	 * ### ROOT VISUALIZATION ###
	 */
	
	bool doRoot = true;
	if (argc > 2) doRoot = (bool)atof(argv[2]);
	
	if (doRoot) {

		TH2D * thatHist = addVectorOfPToHistograms(theHistograms);
		thatHist->GetXaxis()->SetTitle("#alpha / #circ");
		thatHist->GetYaxis()->SetTitle("r / cm");
	
//		for (int i = 0; i < transformedPoints.size(); i++) {
//			for (int j = 0; j < transformedPoints[i].size(); j++) {
//				thatHist->Fill(alphas[j], transformedPoints[i][j]);
//			}
//		}
		
		TApplication *theApp = new TApplication("app", &argc, argv, 0, -1);
		TCanvas * c1 = new TCanvas("c1", "default", 100, 10, 800, 600);
		
		thatHist->Draw("COLZ");
		c1->Update();
		theApp->Run();
	}
	
	
	/*
	 * 
	 * ### TODO ###
	 * 
	 * make peak finder
	 * outline:
	 * 	create rough grid
	 * 		find max of trasnformedPoints via thrust lib to determine borders
	 * 	divide grid
	 * 	find max
	 * 	create fine grid
	 * 	find peaks
	
	
	LINKS FOR MATRIX STUFF:
	
	https://www.google.com/search?q=thrust+matrix
	https://www.google.com/search?q=thrust+matrix+multiplication&sugexp=chrome,mod=10&sourceid=chrome&ie=UTF-8
	http://stackoverflow.com/questions/618511/a-proper-way-to-create-a-matrix-in-c
	http://www.velocityreviews.com/forums/t281152-is-there-any-matrix-in-the-stl.html
	BLAS
	Blitz++
	
	 */
}
